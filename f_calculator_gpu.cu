#include "hip/hip_runtime.h"
#include "f_calculator_gpu.cuh"
#include "ptcl_class.hpp"

//static members for calculating non-bonded force
PS::F64 Parameter::cf_c[Parameter::prop_num][Parameter::prop_num];
PS::F64 Parameter::cf_g[Parameter::prop_num][Parameter::prop_num];
PS::F64 Parameter::cf_r[Parameter::prop_num][Parameter::prop_num];
PS::F64 Parameter::cf_m[Parameter::prop_num][Parameter::prop_num][Parameter::prop_num];

bool Policy::Density::init_call = true;
cuda_ptr<EPI::DensityGPU<VecPos> > Policy::Density::dev_epi;
cuda_ptr<EPJ::DensityGPU<VecPos> > Policy::Density::dev_epj;
cuda_ptr<RESULT::DensityGPU<Dtype> > Policy::Density::dev_force;

bool Policy::Force::init_call = true;
cuda_ptr<EPI::DPDGPU<VecPos> > Policy::Force::dev_epi;
cuda_ptr<EPJ::DPDGPU<VecPos> > Policy::Force::dev_epj;
cuda_ptr<RESULT::ForceGPU<VecForce> > Policy::Force::dev_force;
//

namespace {
#ifdef USE_TEXTURE_MEM
  cuda3D_ptr<Dtype,
    Parameter::prop_num,
    Parameter::prop_num,
    Parameter::prop_num> ptr_cf_m(hipCreateChannelDesc<Dtype>());

  cuda2D_ptr<Dtype,
    Parameter::prop_num,
    Parameter::prop_num> ptr_cf_c(hipCreateChannelDesc<Dtype>()),
                         ptr_cf_r(hipCreateChannelDesc<Dtype>()),
                         ptr_cf_g(hipCreateChannelDesc<Dtype>());
  
#ifdef USE_FLOAT_VEC
  texture<float, hipTextureType3D, hipReadModeElementType> cf_m;
  texture<float, hipTextureType2D, hipReadModeElementType> cf_c;
  texture<float, hipTextureType2D, hipReadModeElementType> cf_r;
  texture<float, hipTextureType2D, hipReadModeElementType> cf_g;
#else
  texture<int2, hipTextureType3D, hipReadModeElementType> cf_m;
  texture<int2, hipTextureType2D, hipReadModeElementType> cf_c;
  texture<int2, hipTextureType2D, hipReadModeElementType> cf_r;
  texture<int2, hipTextureType2D, hipReadModeElementType> cf_g;
#endif //USE_FLOAT_VEC

#else
  __device__ Dtype cf_m[Parameter::prop_num][Parameter::prop_num][Parameter::prop_num];
  __device__ Dtype cf_c[Parameter::prop_num][Parameter::prop_num];
  __device__ Dtype cf_r[Parameter::prop_num][Parameter::prop_num];
  __device__ Dtype cf_g[Parameter::prop_num][Parameter::prop_num];
#endif //USE_TEXTURE_MEM

  enum {
    N_THREAD_GPU = 32,
    N_WALK_LIMIT = 1000,
    NI_LIMIT     = N_WALK_LIMIT*1000,
    NJ_LIMIT     = N_WALK_LIMIT*10000,
  };

  bool gpu_inited = false;
  cuda_ptr<int2> ij_disp;

#ifdef USE_TEXTURE_MEM

  void set_texture_val() {
    ptr_cf_m.host2host(Parameter::cf_m);
    ptr_cf_c.host2host(Parameter::cf_c);
    ptr_cf_r.host2host(Parameter::cf_r);
    ptr_cf_g.host2host(Parameter::cf_g);
    
    ptr_cf_m.host2dev();
    ptr_cf_c.host2dev();
    ptr_cf_r.host2dev();
    ptr_cf_g.host2dev();
    
    cf_m.normalized = false;
    cf_c.normalized = false;
    cf_r.normalized = false;
    cf_g.normalized = false;
    
    hipBindTextureToArray(cf_m, ptr_cf_m.dev_ptr, ptr_cf_m.cdesc);
    hipBindTextureToArray(cf_c, ptr_cf_c.dev_ptr, ptr_cf_c.cdesc);
    hipBindTextureToArray(cf_r, ptr_cf_r.dev_ptr, ptr_cf_r.cdesc);
    hipBindTextureToArray(cf_g, ptr_cf_g.dev_ptr, ptr_cf_g.cdesc);
  }

  void clean_up_texture_mem() {
    ptr_cf_m.deallocate();
    ptr_cf_c.deallocate();
    ptr_cf_r.deallocate();
    ptr_cf_g.deallocate();
  }

#else

  void set_const_gpu() {
    Dtype cf_m_h[Parameter::prop_num][Parameter::prop_num][Parameter::prop_num];
    Dtype cf_c_h[Parameter::prop_num][Parameter::prop_num];
    Dtype cf_r_h[Parameter::prop_num][Parameter::prop_num];
    Dtype cf_g_h[Parameter::prop_num][Parameter::prop_num];
    
    for(int i = 0; i < Parameter::prop_num; i++) {
      for(int j = 0; j < Parameter::prop_num; j++) {
	cf_c_h[i][j] = Parameter::cf_c[i][j];
	cf_r_h[i][j] = Parameter::cf_r[i][j];
	cf_g_h[i][j] = Parameter::cf_g[i][j];
      }
    }

    for(int i = 0; i < Parameter::prop_num; i++)
      for(int j = 0; j < Parameter::prop_num; j++)
	for(int k = 0; k < Parameter::prop_num; k++)
	  cf_m_h[i][j][k] = Parameter::cf_m[i][j][k];
    
    const size_t cf_m_size = sizeof(float) * Parameter::prop_num * Parameter::prop_num * Parameter::prop_num;
    const size_t cf_p_size = sizeof(float) * Parameter::prop_num * Parameter::prop_num;
    void* ptr_dev = NULL;

#define COPY_TO_DEVICE_SYMBOL(dev, host, size) \
    checkCudaErrors(hipGetSymbolAddress(&ptr_dev, HIP_SYMBOL(dev))); \
    checkCudaErrors(hipMemcpy(ptr_dev, host, size, hipMemcpyHostToDevice))
    
    COPY_TO_DEVICE_SYMBOL(cf_m, cf_m_h, cf_m_size);
    COPY_TO_DEVICE_SYMBOL(cf_c, cf_c_h, cf_p_size);
    COPY_TO_DEVICE_SYMBOL(cf_r, cf_r_h, cf_p_size);
    COPY_TO_DEVICE_SYMBOL(cf_g, cf_g_h, cf_p_size);

#undef COPY_TO_DEVICE_SYMBOL
  }

#endif //USE_TEXTURE_MEM

  template<class Policy>
  void clean_up_device_mem() {
    Policy::dev_epi.deallocate();
    Policy::dev_epj.deallocate();
    Policy::dev_force.deallocate();
  }
  
}; //end of anonymous namespace

void clean_up_gpu() {
  ij_disp.deallocate();
  clean_up_device_mem<Policy::Density>();
  clean_up_device_mem<Policy::Force>();

#ifdef USE_TEXTURE_MEM
  clean_up_texture_mem();
#endif
}

#include "kernel_impl.cuh"

template<class Policy, class EPI, class EPJ>
PS::S32 DispatchKernel(const PS::S32 tag,
		       const PS::S32 n_walk,
		       const EPI ** epi,
		       const PS::S32 * n_epi,
		       const EPJ ** epj,
		       const PS::S32 * n_epj) {
  assert(n_walk <= N_WALK_LIMIT);

  //allocate array
  if(!gpu_inited) {
#ifdef USE_TEXTURE_MEM
    set_texture_val();
#else
    set_const_gpu();
#endif
    
    ij_disp.allocate(N_WALK_LIMIT + 2);    
    gpu_inited = true;
  }
  
  if(Policy::init_call) {
    Policy::dev_epi.allocate(NI_LIMIT);
    Policy::dev_epj.allocate(NJ_LIMIT);
    Policy::dev_force.allocate(NI_LIMIT);
    
    Policy::init_call = false;
  }
  
  ij_disp[0].x = ij_disp[0].y = 0;
  for(PS::S32 k = 0; k < n_walk; k++) {
    ij_disp[k + 1].x = ij_disp[k].x + n_epi[k];
    ij_disp[k + 1].y = ij_disp[k].y + n_epj[k];
  }
  ij_disp[n_walk + 1] = ij_disp[n_walk];

  assert(ij_disp[n_walk].x < NI_LIMIT);
  assert(ij_disp[n_walk].y < NJ_LIMIT);
  
  ij_disp.host2dev(0, n_walk + 2);
  
  PS::S32 ni_tot_reg = ij_disp[n_walk].x;
  if(ni_tot_reg % N_THREAD_GPU) {
    ni_tot_reg /= N_THREAD_GPU;
    ni_tot_reg++;
    ni_tot_reg *= N_THREAD_GPU;
  }
  
  PS::S32 ni_tot = -1, nj_tot = -1;
  Policy::CopyToBuffer(n_walk, epi, n_epi, epj, n_epj, ni_tot, nj_tot);
  
  for(PS::S32 i = ni_tot; i < ni_tot_reg; i++)
    Policy::dev_epi[i].id_walk = n_walk;

  Policy::dev_epi.host2dev(0, ni_tot_reg);
  Policy::dev_epj.host2dev(0, nj_tot);
  
  const PS::S32 nblocks = ni_tot_reg / N_THREAD_GPU;
  const PS::S32 nthreads = N_THREAD_GPU;
  ForceKernel<VecPos, VecForce, Dtype> <<< nblocks, nthreads >>> (ij_disp, Policy::dev_epi, Policy::dev_epj, Policy::dev_force, Parameter::time);

  return 0;
}

template<class Policy, class RESULT>
PS::S32 RetrieveKernel(const PS::S32 tag,
		       const PS::S32 n_walk,
		       const PS::S32 ni[],
		       RESULT * force[]) {
  int ni_tot = 0;
  for(int k = 0; k < n_walk; k++)
    ni_tot += ni[k];
  Policy::dev_force.dev2host(0, ni_tot);
  Policy::CopyToOrigin(n_walk, ni, force);
  return 0;
}

//instantiate
template
PS::S32 DispatchKernel<Policy::Density, EPI::Density, EPJ::Density>(const PS::S32 tag,
								    const PS::S32 n_walk,
								    const EPI::Density ** epi,
								    const PS::S32 * n_epi,
								    const EPJ::Density ** epj,
								    const PS::S32 * n_epj);

template
PS::S32 DispatchKernel<Policy::Force,   EPI::DPD,     EPJ::DPD    >(const PS::S32 tag,
								    const PS::S32 n_walk,
								    const EPI::DPD ** epi,
								    const PS::S32 * n_epi,
								    const EPJ::DPD ** epj,
								    const PS::S32 * n_epj);

template
PS::S32 RetrieveKernel<Policy::Density, RESULT::Density> (const PS::S32 tag,
							  const PS::S32 n_walk,
							  const PS::S32 ni[],
							  RESULT::Density * force[]);

template
PS::S32 RetrieveKernel<Policy::Force, RESULT::ForceDPD> (const PS::S32 tag,
							 const PS::S32 n_walk,
							 const PS::S32 ni[],
							 RESULT::ForceDPD * force[]);

