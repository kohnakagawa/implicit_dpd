#include "hip/hip_runtime.h"
#include <stdio.h>

enum {
  Nx = 3,
  Ny = 4,
  Nz = 5,
};

texture<float, hipTextureType2D, hipReadModeElementType> texture2D;
texture<float, hipTextureType3D, hipReadModeElementType> texture3D;
texture<int2, hipTextureType2D, hipReadModeElementType> texture2D_d;

__global__ void kernel2D() {
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if(tid == 0) {
    for(int ix = 0; ix < Nx; ix++) {
      for(int iy = 0; iy < Ny; iy++) {
	const float val = tex2D(texture2D, iy, ix);
	const int2 v = tex2D(texture2D_d, iy, ix);
	const double val_d = __hiloint2double( v.y, v.x );
	printf("%f %f\n", val, val_d);
      }
    }
  }
}

__global__ void kernel3D() {
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if(tid == 0) {
    for(int ix = 0; ix < Nx; ix++) {
      for(int iy = 0; iy < Ny; iy++) {
	for(int iz = 0; iz < Nz; iz++) {
	  const float val = tex3D(texture3D, iz, iy, ix);
	  printf("%f\n", val);
	}
      }
    }
  }  
}

int main() {
  float host_2D[Nx][Ny]; //H W
  float host_3D[Nx][Ny][Nz]; //D H W
  double host_2D_d[Nx][Ny]; //H W

  int cnt = 0;
  for(int ix = 0; ix < Nx; ix++) {

  for(int iy = 0; iy < Ny; iy++) {
      host_2D[ix][iy] = cnt;
      host_2D_d[ix][iy] = cnt;
      cnt++;
    }
  }
  cnt = 0;
  for(int ix = 0; ix < Nx; ix++) {
    for(int iy = 0; iy < Ny; iy++) {
      for(int iz = 0; iz < Nz; iz++) {
        host_3D[ix][iy][iz] = cnt;
        cnt++;
      }
    }
  }
  
  hipArray *cu_2D = nullptr, *cu_3D = nullptr, *cu_2D_d = nullptr;
  hipChannelFormatDesc cdesc = hipCreateChannelDesc<float>();
  hipChannelFormatDesc cdesc_d = hipCreateChannelDesc<int2>();
  
  hipMallocArray(&cu_2D, &cdesc, Ny, Nx);
  hipMalloc3DArray(&cu_3D, &cdesc, make_hipExtent(Nz, Ny, Nx) );
  hipMallocArray(&cu_2D_d, &cdesc_d, Ny, Nx);

  const size_t size2d = Nx * Ny * sizeof(float);
  hipMemcpyToArray(cu_2D, 0, 0, host_2D, size2d, hipMemcpyHostToDevice);

  const size_t size2d_d = Nx * Ny * sizeof(double);
  hipMemcpyToArray(cu_2D_d, 0, 0, host_2D_d, size2d_d, hipMemcpyHostToDevice);

  hipMemcpy3DParms copyParams = {0};
  copyParams.srcPtr = make_hipPitchedPtr(host_3D, Nz * sizeof(float), Nz, Ny);
  copyParams.dstArray = cu_3D;
  copyParams.extent = make_hipExtent(Nz, Ny, Nx); //width height depth
  copyParams.kind   = hipMemcpyHostToDevice;
  hipMemcpy3D(&copyParams);

  texture2D.normalized = false;
  texture3D.normalized = false;
  
  hipBindTextureToArray(texture2D, cu_2D, cdesc);
  hipBindTextureToArray(texture3D, cu_3D, cdesc);
  hipBindTextureToArray(texture2D_d, cu_2D_d, cdesc_d);
  
  kernel2D<<<32, 10>>>();
  kernel3D<<<32, 10>>>();

  hipDeviceSynchronize();
  
  hipFreeArray(cu_2D);
  hipFreeArray(cu_3D);
}